﻿#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "Headers/file.cuh"

struct GenRandInt {
	__device__
		int operator () (int idx) {
		thrust::default_random_engine rand_eng;
		thrust::uniform_int_distribution<int> uni_dist;
		rand_eng.discard(idx);
		return uni_dist(rand_eng);
	}
};


bool fileExists(const std::string& name) {
	struct stat buffer;
	return stat(name.c_str(), &buffer) == 0;
}

int main(int argc, char* argv[]) {
	size_t N = 3;
	if(argc != 3) {
		std::cout << "Requires 2 args\n";
		return 0;
	}
	else {
		N = std::atoi(argv[2]);
		if(N < 2 || N > 30000) {
			std::cout << "Invalid size parameter (2 <= size <= 30000).\n";
			return 0;
		}
	}
	std::string fileName = argv[1];
	if(!fileExists(fileName)) {
		std::cout << "File does not exist!\n";
		return 0;
	}
	const size_t size = N * N;
	thrust::device_vector<int> a(size);
	thrust::transform(thrust::make_counting_iterator(0ULL), thrust::make_counting_iterator(size), a.begin(), GenRandInt());
	thrust::host_vector<int> a_copy = a;
	SaveToBinary(a_copy, fileName);
	return 0;
}