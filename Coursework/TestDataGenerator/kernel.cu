﻿#include "hip/hip_runtime.h"
#include <thrust/random.h>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "Headers/file.cuh"

struct GenRandInt {
	__device__
		int operator () (int idx) {
		thrust::default_random_engine rand_eng;
		thrust::uniform_int_distribution<int> uni_dist;
		rand_eng.discard(idx);
		return uni_dist(rand_eng);
	}
};

const std::string base_dir = "D:\\University\\Courseworks\\AVS\\AVS-course-project\\Coursework\\TestDataGenerator\\data\\";


int main() {
	const size_t N = 30000;
	const size_t size = N * N;
	thrust::device_vector<int> a(size);
	thrust::transform(
   thrust::make_counting_iterator(0ULL),
   thrust::make_counting_iterator(size),
   a.begin(),
   GenRandInt());

	thrust::host_vector<int> a_copy = a;

	SaveToTxt(a_copy, base_dir + "3e1_int.txt");
	//SaveToBinary(a_copy, base_dir + "3e1_int.dat");
	//thrust::host_vector<int> v;
	//ReadFromBinary(v, base_dir + "3e1_int.dat");
	return 0;
}