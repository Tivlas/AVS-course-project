#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <chrono>
#include <fstream>
#include <atomic>
#include <thread>
#include <functional>

template <typename T>
__global__ void matrixMulKernel(T* c, const T* a, const T* b, size_t N) {
	for(int i = 0; i < N; i++) {
		for(int k = 0; k < N; k++) {
			for(int j = 0; j < N; j++) {
				c[i * N + j] += a[i * N + k] * b[k * N + j];
			}
		}
	}
}

template <typename T>
__global__ void matrixAddKernel(T* c, const T* a, const T* b, size_t N) {
	N *= N;
	for(int i = 0; i < N; i++) {
		c[i] = a[i] + b[i];
	}
}

template <typename T>
void ReadFromBinary(thrust::host_vector<T>& v, const std::string& filename) {
	std::ifstream file(filename, std::ios::binary);
	if(!file.good()) throw std::invalid_argument("Invalid file!");
	size_t size = 0;
	file.read(reinterpret_cast<char*>(&size), sizeof(size_t));
	v.resize(size);
	file.read(reinterpret_cast<char*>(v.data()), size * sizeof(T));
	file.close();
}

void printProgressIndicator(std::atomic<bool>& isCalculating) {
	const std::string indicators = "-\\|/";
	int index = 0;
	while(isCalculating) {
		std::cout << "\rCalculating... " << indicators[index++];
		index %= indicators.size();
		std::cout.flush();
		std::this_thread::sleep_for(std::chrono::milliseconds(200));
	}
	std::cout << '\n';
}

template <typename T>
auto measureTime(const thrust::device_vector<T>& a, thrust::device_vector<T>& b, thrust::device_vector<T>& result, size_t N, std::string op) {
	std::atomic<bool> isCalculating;
	isCalculating.store(true);
	std::thread progressThread(printProgressIndicator, std::ref(isCalculating));
	std::this_thread::sleep_for(std::chrono::milliseconds(500));

	auto start = std::chrono::high_resolution_clock::now();
	if(op == "m") {
		matrixMulKernel << <1, 1 >> > (thrust::raw_pointer_cast(result.data()), thrust::raw_pointer_cast(a.data()), thrust::raw_pointer_cast(b.data()), N);
	}
	else if(op == "a") {
		matrixAddKernel << <1, 1 >> > (thrust::raw_pointer_cast(result.data()), thrust::raw_pointer_cast(a.data()), thrust::raw_pointer_cast(b.data()), N);
	}
	hipDeviceSynchronize();
	auto end = std::chrono::high_resolution_clock::now();
	auto elapsed_time = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start);
	isCalculating.store(false);
	progressThread.join();
	return elapsed_time.count();
}

template <typename T>
void calculate(std::string op, const std::string& fileName) {
	thrust::host_vector<T> h_a;
	try {
		ReadFromBinary<T>(h_a, fileName);
	}
	catch(const std::exception& e) {
		std::cout << e.what();
		return;
	}
	thrust::device_vector<T> d_a = h_a;
	thrust::device_vector<T> d_b = h_a;
	thrust::device_vector<T> d_c(h_a.size());
	const size_t N = sqrt(h_a.size());
	auto time = measureTime<T>(d_a, d_b, d_c, N, op);
	std::cout << time << '\n';
}

int main(int argc, char* argv[]) {
	if(argc != 4) {
		std::cout << "Requires 3 args\n";
		return 0;
	}
	else {
		std::string op = argv[2];
		if(op != "a" && op != "m") {
			std::cout << "Usage: <file_path> <function (m or a)>\n";
			return 0;
		}
		std::string dataType = argv[3];
		if(dataType != "i" && dataType != "f") {
			std::cout << "Usage: <file_path> <function (m or a)> <dataType (i or f)\n";
			return 0;
		}
	}
	std::string fileName = argv[1];
	std::string op = argv[2];
	std::string dataType = argv[3];
	if(dataType == "i") {
		calculate<int>(op, fileName);
	}
	else if(dataType == "f") {
		calculate<float>(op, fileName);
	}
	return 0;
}